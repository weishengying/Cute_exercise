#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        std::cout << "Device " << i << ":\n";
        std::cout << "  Name: " << prop.name << "\n";
        std::cout << "  Registers per block: " << prop.regsPerBlock << "\n";
        std::cout << "  Registers per SM: " << prop.regsPerMultiprocessor << "\n";
        std::cout << "  Registers per SM: " << prop.regsPerMultiprocessor * 4 << " bytes\n";
        std::cout << "  Shared memory per block: " << prop.sharedMemPerBlock << " bytes\n";
        std::cout << "  Shared memory per SM: " << prop.sharedMemPerMultiprocessor << " bytes\n";
    }

    return 0;
}
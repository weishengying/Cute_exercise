#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <float.h>
#include "utils.h"

using T = cute::half_t;
using namespace cute;

template <typename T, int BM, int BN, int BK, typename TiledMMA, 
            typename G2SCopyA, typename G2SCopyB,
            typename SmemLayoutA, typename SmemLayoutB,
            typename S2RCopyAtomA, typename S2RCopyAtomB>
__global__ void gemm_v2_kernel(const T *Aptr, const T *Bptr, T *Dptr, int m, int n, int k) {
    // Initilize shared memory
    extern __shared__ T shm_data[];
    T *Ashm = shm_data;
    T *Bshm = shm_data + cute::cosize(SmemLayoutA{});

    // Initilize thread block
    int idx = threadIdx.x;
    int ix = blockIdx.x;
    int iy = blockIdx.y;

    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor D = make_tensor(make_gmem_ptr(Dptr), make_shape(m, n), make_stride(n, Int<1>{}));

    // Global Memory
    Tensor gA = local_tile(A, make_tile(Int<BM>{}, Int<BK>{}), make_coord(iy, _)); // (BM, BK, num_tile_k)
    Tensor gB = local_tile(B, make_tile(Int<BN>{}, Int<BK>{}), make_coord(ix, _)); // (BN, BK, num_tile_k)
    Tensor gD = local_tile(D, make_tile(Int<BM>{}, Int<BN>{}), make_coord(iy, ix)); // (BM, BN) 

    // shared memory
    auto sA = make_tensor(make_smem_ptr(Ashm), SmemLayoutA{}); // (BM, BK)
    auto sB = make_tensor(make_smem_ptr(Bshm), SmemLayoutB{}); // (BN, BK)

    // register, use tiled_mma to partition register A/B/C
    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    auto tCgD = thr_mma.partition_C(gD); // (MMA, MMA_M, MMA_N)

    auto tCrA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
    auto tCrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrD = thr_mma.partition_fragment_C(gD);           // (MMA, MMA_M, MMA_N)
    clear(tCrD);

    // from global memory to shared memory
    G2SCopyA g2s_tiled_copy_a;
    auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
    auto tAgA_copy = g2s_thr_copy_a.partition_S(gA); // (CPY, CPY_M, CPY_K, num_tile_k)
    auto tAsA_copy = g2s_thr_copy_a.partition_D(sA); // (CPY, CPY_M, CPY_K)

    G2SCopyB g2s_tiled_copy_b;
    auto g2s_thr_copy_b = g2s_tiled_copy_b.get_slice(idx);
    auto tBgB_copy = g2s_thr_copy_b.partition_S(gB); // (CPY, CPY_N, CPY_K, num_tile_k)
    auto tBsB_copy = g2s_thr_copy_b.partition_D(sB); // (CPY, CPY_N, CPY_K)

    // from shared memory to register, use tiled_mma to generate tiled_copy
    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, tiled_mma);
    auto s2r_thr_copy_a = s2r_tiled_copy_a.get_slice(idx);
    auto tAsA = s2r_thr_copy_a.partition_S(sA);     // (CPY, CPY_M, CPY_K)
    auto tCrA_view = s2r_thr_copy_a.retile_D(tCrA); // (CPY, CPY_M, CPY_K) 这里的 CPY_M 和 CPY_K 与上面 G2SCopyA 的 CPY_M、CPY_K 并不相同

    auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, tiled_mma);
    // auto s2r_thr_copy_b = s2r_tiled_copy_b.get_slice(idx);
    // auto tBsB = s2r_thr_copy_b.partition_S(sB);     // (CPY, CPY_N, CPY_K)
    // auto tCrB_view = s2r_thr_copy_b.retile_D(tCrB); // (CPY, CPY_N, CPY_K)


    if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    {
        PRINT("tCrA", tCrA.shape())    
        // PRINT("tCrB", tCrB.shape())   
        
        PRINT("tAgA_copy", tAgA_copy.shape())      
        // PRINT("tBgB_copy", tBgB_copy.shape())     
        // print_latex(s2r_tiled_copy_a);
        PRINT("tAsA", tAsA.shape())     
        PRINT("tCrA_view", tCrA_view.shape()) 
        // // print(layout<0>(tBgB));
        // // PRINT("tBrB", tBrB.shape()) 

        // PRINT("tBsB", tBsB.shape())     
        // PRINT("tCrB_view", tCrB_view.shape()) 
    }
}

template <typename T>
void gemm_v2(const T *a, const T *b, T *c, int M, int N, int K) {
    auto BM = Int<128>{};
    auto BN = Int<256>{};
    auto BK = Int< 32>{};


    // Define the smem layouts
    using SmemLayoutAtom = decltype(composition(
        Swizzle<3, 3, 3>{},
        make_layout(make_shape(Int<8>{}, Int<BK>{}),
                    make_stride(Int<BK>{}, Int<1>{}))));
    using SmemLayoutA = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BM>{}, Int<BK>{})));
    using SmemLayoutB = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BN>{}, Int<BK>{})));
    
    // copy from global memory to shared memory
    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>;
    using G2SCopyA =
        decltype(make_tiled_copy(g2s_copy_atom{},
                                 make_layout(make_shape(Int<8>{}, Int<4>{}), // Thr layout 32x4 k-major
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{})))); // Val layout 1x8
    using G2SCopyB = G2SCopyA;

    // mma
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int kMmaEURepeatM = 1;
    static constexpr int kMmaEURepeatN = 1;
    static constexpr int kMmaEURepeatK = 1;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 16 * kMmaEURepeatM;
    static constexpr int kMmaPN = 8 * kMmaEURepeatN * 2;
    static constexpr int kMmaPK = 16 * kMmaEURepeatK;
    using MMA_EU_RepeatT = decltype(make_layout(make_shape(
        Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));


    // copy from shared memory to register
    // use mma tiled ,so no tiled here
    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
    using S2RCopyAtomA = s2r_copy_atom;
    using S2RCopyAtomB = s2r_copy_atom;
    
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    dim3 block(size(MMA{}));
    dim3 grid(BX, BY);

    gemm_v2_kernel<T, BM, BN, BK, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, S2RCopyAtomA, S2RCopyAtomB>
               <<<grid, block>>>(a, b, c, M, N, K);
}


int main() {

    const int test_num = 1;
    int M_list[test_num];
    int N_list[test_num];
    int K_list[test_num];

    for (int i = 0; i < test_num; i++) {
        M_list[i] = (i + 1) * 256;
        N_list[i] = (i + 1) * 256;
        K_list[i] = (i + 1) * 256;
    }

    for (int j = 0; j < test_num; j++) {
        int M = M_list[j], N = N_list[j], K = K_list[j];
        size_t size_a = M * K * sizeof(T);
        size_t size_b = K * N * sizeof(T);
        size_t size_c = M * N * sizeof(T);

        T *d_a, *d_b;
        T *d_c;
        hipMalloc(&d_a, size_a);
        hipMalloc(&d_b, size_b);
        hipMalloc(&d_c, size_c);
        gemm_v2(d_a, d_b, d_c, M, N, K);
        hipDeviceSynchronize();
    }

    return 0;
}